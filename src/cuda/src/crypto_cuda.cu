#include "hip/hip_runtime.h"
#include "crypto.cpp"

template <typename T>
__global__ void bd09_to_gcj02_cuda(T *lon,
                                   T *lat)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    bd09_to_gcj02(lon[i], lat[i], lon[i], lat[i]);
};

template <typename T>
__global__ void gcj02_to_bd09_cuda(T *lon,
                                   T *lat)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    gcj02_to_bd09(lon[i], lat[i], lon[i], lat[i]);
};
template <typename T>
__global__ void gcj02_to_wgs84_cuda(T *lon,
                                    T *lat)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    gcj02_to_wgs84(lon[i], lat[i], lon[i], lat[i]);
};
template <typename T>
__global__ void wgs84_to_gcj02_cuda(T *lon,
                                    T *lat)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    wgs84_to_gcj02(lon[i], lat[i], lon[i], lat[i]);
};
template <typename T>
__global__ void wgs84_to_bd09_cuda(T *lon,
                                   T *lat)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    wgs84_to_bd09(lon[i], lat[i], lon[i], lat[i]);
};
template <typename T>
__global__ void bd09_to_wgs84_cuda(T *lon,
                                   T *lat)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    bd09_to_wgs84(lon[i], lat[i], lon[i], lat[i]);
};

template <typename T>
__global__ void gcj02_to_wgs84_exact_cuda(T *lon,
                                          T *lat,
                                          const T threshold,
                                          const bool distance_mode,
                                          const int max_iter)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    crypto_exact(lon[i], lat[i], gcj02_to_wgs84, wgs84_to_gcj02, threshold, distance_mode, max_iter, lon[i], lat[i]);
};
template <typename T>
__global__ void bd09_to_wgs84_exact_cuda(T *lon,
                                         T *lat,
                                         const T threshold,
                                         const bool distance_mode,
                                         const int max_iter)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    crypto_exact(lon[i], lat[i], bd09_to_wgs84, wgs84_to_bd09, threshold, distance_mode, max_iter, lon[i], lat[i]);
};
template <typename T>
__global__ void bd09_to_gcj02_exact_cuda(T *lon,
                                         T *lat,
                                         const T threshold,
                                         const bool distance_mode,
                                         const int max_iter)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    crypto_exact(lon[i], lat[i], bd09_to_gcj02, gcj02_to_bd09, threshold, distance_mode, max_iter, lon[i], lat[i]);
};
